#include <stdio.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA_ERROR(file, line) {                                  \
    hipError_t err_t;                                                  \
    if ((err_t = hipGetLastError() ) != hipSuccess) {                 \
      printf("Cuda error: %s \n", hipGetErrorString(err_t));             \
      printf("File: %s; line %d\n", file, line);                          \
      exit(1);                                                          \
    }                                                                   \
  }   


const int N = 1000000; // 1M
const int blocksize = 16;

__global__
void vupdate(const int n, const int *x, int *y)
{

  int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < n)
      y[i] = x[i] + y[i];

}

int main()
{
  int *h_x, *h_y;
  int *d_x, *d_y;

  hipEvent_t start, stop;
  float elapsedTime;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  // Allocating on the CPU
  h_x =  (int *) malloc(N*sizeof(int));
  h_y =  (int *) malloc(N*sizeof(int));

  // init data
  for (int i=0; i<N; ++i) {
    h_x[i] = i;
    h_y[i] = i-10;
  }

  // Allocating on the GPU
  hipMalloc( (void**)&d_x, N*sizeof(int) );
  CHECK_CUDA_ERROR(__FILE__, __LINE__);
  hipMalloc( (void**)&d_y, N*sizeof(int) );
  CHECK_CUDA_ERROR(__FILE__, __LINE__);

  // Copy data from CPU to GPU
  hipMemcpy( d_x, h_x, N*sizeof(int), hipMemcpyHostToDevice );
  CHECK_CUDA_ERROR(__FILE__, __LINE__);
  hipMemcpy( d_y, h_y, N*sizeof(int), hipMemcpyHostToDevice );
  CHECK_CUDA_ERROR(__FILE__, __LINE__);

  dim3 dimBlock(blocksize);
  dim3 dimGrid(N / blocksize + 1);

  for (int test=0; test<500; ++test) {
    vupdate<<<dimGrid, dimBlock>>>(N, d_x, d_y);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);
  }

  // copy back
  hipMemcpy( h_x, d_x , N*sizeof(int), hipMemcpyDeviceToHost );
  CHECK_CUDA_ERROR(__FILE__, __LINE__);
  hipMemcpy( h_y, d_y , N*sizeof(int), hipMemcpyDeviceToHost );
  CHECK_CUDA_ERROR(__FILE__, __LINE__);

  // Free on GPU
  hipFree( d_x );
  CHECK_CUDA_ERROR(__FILE__, __LINE__);
  hipFree( d_y );
  CHECK_CUDA_ERROR(__FILE__, __LINE__);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipEventElapsedTime(&elapsedTime, start, stop);

  printf ("Time for the kernel: %f ms\n", elapsedTime);

  // Free on host
  free(h_x);
  free(h_y);

  return 0;
}
